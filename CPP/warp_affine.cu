#include "hip/hip_runtime.h"
#include "warp_affine.h"
#include <hip/hip_runtime.h>
#include <math.h>

#define min(a, b)  ((a) < (b) ? (a) : (b))

void AffineMatrix::invertAffineTransform(float imat[6], float omat[6]){
    float i00 = imat[0];  float i01 = imat[1];  float i02 = imat[2];
    float i10 = imat[3];  float i11 = imat[4];  float i12 = imat[5];
    float D = i00 * i11 - i01 * i10;
    D = D != 0 ? 1.0 / D : 0;
    float A11 = i11 * D;
    float A22 = i00 * D;
    float A12 = -i01 * D;
    float A21 = -i10 * D;
    float b1 = -A11 * i02 - A12 * i12;
    float b2 = -A21 * i02 - A22 * i12;
    omat[0] = A11;  omat[1] = A12;  omat[2] = b1;
    omat[3] = A21;  omat[4] = A22;  omat[5] = b2;
}

void AffineMatrix::compute(const Size& from, const Size& to){
    float scale_x = to.width / (float)from.width;
    float scale_y = to.height / (float)from.height;
    float scale = min(scale_x, scale_y);
    i2d[0] = scale;  i2d[1] = 0;  i2d[2] =
        -scale * from.width  * 0.5  + to.width * 0.5 + scale * 0.5 - 0.5;
    i2d[3] = 0;  i2d[4] = scale;  i2d[5] =
        -scale * from.height * 0.5 + to.height * 0.5 + scale * 0.5 - 0.5;
    invertAffineTransform(i2d, d2i);
}

__device__ void affine_project(float* matrix, int x, int y, float* proj_x, float* proj_y){
    *proj_x = matrix[0] * x + matrix[1] * y + matrix[2];
    *proj_y = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__ void warp_affine_bilinear_kernel(
    uint8_t* src, int src_line_size, int src_width, int src_height,
    float* dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value, AffineMatrix matrix
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dx = idx % dst_width;
    const int dy = idx / dst_width;
    if (dx >= dst_width || dy >= dst_height)
        return;
    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0; float src_y = 0;
    affine_project(matrix.d2i, dx, dy, &src_x, &src_y);
    if(src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height){
        c0 = fill_value;
        c1 = fill_value;
        c2 = fill_value;
    }else{
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;
        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_values;
        uint8_t* v2 = const_values;
        uint8_t* v3 = const_values;
        uint8_t* v4 = const_values;
        if(y_low >= 0){
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;
            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }
        if(y_high < src_height){
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;
            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }
        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }
    c0 = (c0-0)/255.0;
    c1 = (c1-0)/255.0;
    c2 = (c2-0)/255.0;
    int stride = dst_width*dst_height;
    dst[dy*dst_width + dx] = c2;
    dst[stride + dy*dst_width + dx] = c1;
    dst[stride*2 + dy*dst_width + dx] = c0;
}

float warp_affine_bilinear(
    uint8_t* src, int src_line_size, int src_width, int src_height,
    float* dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value, hipStream_t stream
){
    const int n = dst_width*dst_height;
    int block_size = 1024;
    const int grid_size = (n + block_size - 1) / block_size;
    AffineMatrix affine;
    affine.compute(Size(src_width, src_height), Size(dst_width, dst_height));
    warp_affine_bilinear_kernel<<<grid_size, block_size, 0, stream>>>(
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value, affine
    );
    return affine.i2d[0];
}